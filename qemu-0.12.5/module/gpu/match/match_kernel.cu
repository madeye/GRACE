#include "hip/hip_runtime.h"
#ifndef _HISTORY_KERNEL_H_
#define _HISTORY_KERNEL_H_

__constant__ uint8_t d_max_tid_num;

#include "../../cpu/detector/race.c"
#include "../../cpu/detector/timestamp.c"
#include "../../cpu/detector/history.c"
#include "../../cpu/detector/filter.c"

__global__ static void module_match_with_trace_buf_on_cuda(
        uint32_t size, struct trace_content *trace_buf)
{
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    struct trace_content *content = &trace_buf[i];

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_match(content);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_match(content);
    }
}

#endif /* _HISTORY_KERNEL_H_ */
