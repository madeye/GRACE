#include "hip/hip_runtime.h"
#ifndef _HISTORY_KERNEL_H_
#define _HISTORY_KERNEL_H_

__constant__ uint8_t d_max_tid_num;

#include "../../cpu/detector/race.c"
#include "../../cpu/detector/timestamp.c"
#include "../../cpu/detector/history.c"
#include "../../cpu/detector/filter.c"

__global__ static void module_match_with_trace_buf_on_cuda(
        uint32_t size, struct trace_content *trace_buf)
{
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    /*__shared__ trace_content content[blockDim.x];*/

    if (i >= size)
        return;

    /*content[threadIdx.x] = trace_buf[i];*/

    struct trace_content *content = &trace_buf[i];
    
    /*struct trace_content *ptr = content + 32;*/
    /*asm volatile ("prefetch.L1 [%0];"::"l"(ptr));*/

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_match(content);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_match(content);
    }
}

#endif /* _HISTORY_KERNEL_H_ */
