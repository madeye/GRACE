#include "hip/hip_runtime.h"
#ifndef _HISTORY_KERNEL_H_
#define _HISTORY_KERNEL_H_

__device__ struct global_timestamp_queue gts;
__device__ struct global_history_queue gh;
__device__ struct global_page_filter gpf;
__device__ struct global_race_queue gr;

__constant__ int d_max_tid_num;

__device__ static inline int module_timestamp_order_on_cuda(
        uint8_t tid1, uint32_t index1, uint8_t tid2, uint32_t index2) 
{
    const struct timestamp *ts1 = &gts.thread[tid1].entry[index1];
    const struct timestamp *ts2 = &gts.thread[tid2].entry[index2];

    return (((ts1->scalar[tid1] < ts2->scalar[tid1]) && 
                (ts1->scalar[tid2] < ts2->scalar[tid2])) || 
            ((ts1->scalar[tid1] > ts2->scalar[tid1]) && 
             (ts1->scalar[tid2] > ts2->scalar[tid2])));
}

__device__ static inline int module_race_content_equal(
        struct trace_content *content1, struct trace_content *content2)
{
    if ((content1->type == content2->type) && 
            (content1->size == content2->size) && 
            (content1->pc == content2->pc)) {
        return 1;
    }

    return 0;
}


__device__ static inline void module_race_collection_on_cuda(
        struct trace_content *content1, struct trace_content *content2)
{
    uint32_t i;
    struct race_queue *temp_queue;

    temp_queue = &gr.thread[0];

    for (i = 0; i < temp_queue->count; i++) {
        if (module_race_content_equal(content1, 
                    &temp_queue->entry[i].content1) && 
                module_race_content_equal(content2, 
                    &temp_queue->entry[i].content2)) {
            temp_queue->entry[i].instance++;

            break;
        }
    }

    if (i >= temp_queue->count) {
        memcpy(&temp_queue->entry[i].content1, 
                content1, sizeof(struct trace_content));
        memcpy(&temp_queue->entry[i].content2, 
                content2, sizeof(struct trace_content));
        temp_queue->entry[i].instance++;

        temp_queue->count++;
    }

    gr.sum++;
}

__device__ static inline void module_match_with_load_on_cuda(
        struct trace_content *content, const uint8_t other_tid)
{
    uint64_t other_address;
    uint32_t other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    const uint8_t tid = content->tid;
    const uint64_t address = content->address;
    const uint32_t index = content->index;

    temp_queue = &gh.thread[other_tid].hash[(
            address >> HASH_BASE_BIT) % MAX_HASH_NUM];

    tail = temp_queue->load_tail;
    head = tail + 1;
    if (head == MAX_LOAD_QUEUE_SIZE) {
        head = 0;
    }

    /*last_index = gts.thread[other_tid].count;*/
    last_index = MAX_TIMESTAMP_NUM;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_LOAD_QUEUE_SIZE;
        }
        tail--;

        temp_entry= &temp_queue->load_entry[tail];
        other_index = temp_entry->content.index;

        if (last_index != other_index) {
            if (module_timestamp_order_on_cuda(
                        other_tid, other_index, tid, index)) {
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            module_race_collection_on_cuda(&temp_entry->content, content);

            break;
        }
    }
}

__device__ static inline void module_match_with_store_on_cuda(
        struct trace_content *content, const uint8_t other_tid)
{
    uint64_t other_address;
    uint32_t other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    const uint8_t tid = content->tid;
    const uint64_t address = content->address;
    const uint32_t index = content->index;

    temp_queue = &gh.thread[other_tid].hash[(
            address >> HASH_BASE_BIT) % MAX_HASH_NUM];

    tail = temp_queue->store_tail;
    head = tail + 1;
    if (head == MAX_STORE_QUEUE_SIZE) {
        head = 0;
    }

    /*last_index = gts.thread[other_tid].count;*/
    last_index = MAX_TIMESTAMP_NUM;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_STORE_QUEUE_SIZE;
        }
        tail--;

        temp_entry = &temp_queue->store_entry[tail];
        other_index = temp_entry->content.index;

        if (last_index != other_index) {
            if (module_timestamp_order_on_cuda(
                        other_tid, other_index, tid, index)) {
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            module_race_collection_on_cuda(&temp_entry->content, content);

            break;
        }
    }
}

__device__ static inline void module_filter_load_before_match_on_cuda(
        struct trace_content *content)
{
    const uint8_t tid = content->tid;
    const uint64_t address = content->address;
    const uint32_t index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (uint8_t i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (gpf.thread[i].entry[index].store) {
                module_match_with_store_on_cuda(content, i);
            }
        }
    }
}


__device__ static inline void module_filter_store_before_match_on_cuda(
        struct trace_content *content)
{
    const uint8_t tid = content->tid;
    const uint64_t address = content->address;
    const uint32_t index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (uint8_t i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (gpf.thread[i].entry[index].load) {
                module_match_with_load_on_cuda(content, i);
            }

            if (gpf.thread[i].entry[index].store) {
                module_match_with_store_on_cuda(content, i);
            }
        }
    }
}

__global__ static void module_match_with_trace_buf_on_cuda_r(
        int size, struct trace_content *trace_buf)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;

    struct trace_content *content = &trace_buf[i];
	module_filter_load_before_match_on_cuda(content);
}

__global__ static void module_match_with_trace_buf_on_cuda_w(
        int size, struct trace_content *trace_buf)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;

    struct trace_content *content = &trace_buf[i];

    module_filter_store_before_match_on_cuda(content);

}
__global__ static void module_match_with_trace_buf_on_cuda(
        int size, struct trace_content *trace_buf)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;

    struct trace_content *content = &trace_buf[i];

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_before_match_on_cuda(content);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_before_match_on_cuda(content);
    }
}

#endif /* _HISTORY_KERNEL_H_ */
