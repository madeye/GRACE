#include <stdint.h>

#include <cutil_inline.h>

#include "../interface.h"
#include "data.h"
#include "match.h"
#include "match_kernel.cu"

int numThreads = 256;
int numBlocks = 1024;

struct trace_content *d_trace_buf;
#ifdef PPI_THREE_STAGE
struct trace_content *cuda_buf;
#endif

uint32_t old_index[MAX_PROCESS_NUM];

__host__ void module_cuda_tid_update_interface(int h_max_tid_num) {

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_max_tid_num), &h_max_tid_num, 
                sizeof(int)));

}

__host__ void module_cuda_config_register(int _numThreads)
{
    numThreads = _numThreads;
#if 1
    printf("\ncuda : thread number for each block : %d\n", numThreads);
#endif
}

#ifdef KERNEL_TIME
    hipEvent_t start;
    hipEvent_t stop;
    float cuda_time = 0.f;
#endif

__host__ void module_cuda_init_interface()
{
    hipSetDevice(cutGetMaxGflopsDeviceId());

#ifdef KERNEL_TIME
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    memset (old_index, 0, MAX_PROCESS_NUM * sizeof(uint32_t));

#define TRACE_CUDA_BUF_SIZE (16 * 1024 * 1024)

    cutilSafeCall(hipMalloc((void **)&d_trace_buf, 
                sizeof(struct trace_content) * TRACE_CUDA_BUF_SIZE));

#ifdef PPI_THREE_STAGE
    cutilSafeCall(hipHostAlloc((void **)&cuda_buf, sizeof(struct trace_content) * TRACE_BUF_CUDA_SIZE * 2, hipHostMallocDefault));
#else
    cuda_buf = (struct trace_content *) malloc (sizeof(struct trace_content) *
            TRACE_BUF_SIZE * 2);
#endif

#if 1
    printf("\ncuda : global timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));
    printf("cuda : global history queue size : 0x%lx\n", 
            sizeof(struct global_history_queue));
    printf("cuda : global page filter size : 0x%lx\n", 
            sizeof(struct global_page_filter));
    printf("cuda : global race queue size : 0x%lx\n", 
            sizeof(struct global_race_queue));
#endif
}

__host__ void module_cuda_free_interface()
{
    cutilSafeCall(hipFree(d_trace_buf));
    cutilSafeCall(hipHostFree(cuda_buf));

    printf("CUDA Time: %.2fs\n", cuda_time / 1000.0);
}

__host__ void module_cuda_global_timestamp_queue_update_interface(
        struct global_timestamp_queue *h_gts_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_gts_queue, 
                sizeof(struct global_timestamp_queue)));
}

__host__ void module_cuda_global_timestamp_queue_fetch_interface(
        struct global_timestamp_queue *h_gts_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_gts_queue, HIP_SYMBOL(gts), 
                sizeof(struct global_timestamp_queue)));
}

__host__ void module_cuda_timestamp_queue_update_interface(
        uint8_t tid, struct timestamp_queue *h_ts_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_ts_queue, 
                sizeof(struct timestamp_queue), 
                tid * sizeof(struct timestamp_queue), 
                hipMemcpyHostToDevice));
}

__host__ void module_cuda_timestamp_queue_fetch_interface(
        uint8_t tid, struct timestamp_queue *h_ts_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_ts_queue, HIP_SYMBOL(gts), 
                sizeof(struct timestamp_queue), 
                tid * sizeof(struct timestamp_queue), 
                hipMemcpyDeviceToHost));
}

__host__ void module_cuda_timestamp_entry_update_interface_old(
        uint8_t tid, uint32_t index, struct timestamp *h_ts_entry)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_ts_entry, 
                sizeof(struct timestamp),
                tid * sizeof(struct timestamp_queue) +
                index * sizeof(struct timestamp),
                hipMemcpyHostToDevice));
}

#if 1
__host__ void module_cuda_timestamp_entry_update_interface(
        uint8_t max_tid_num, uint32_t *ctx, struct timestamp_queue *h_ts_queue)
{
    int i, index;
    for (i = 1; i < max_tid_num; i++) {
        /*for (i = 0; i < MAX_PROCESS_NUM; i++) {*/

        /*cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry, */
        /*(MAX_TIMESTAMP_NUM) * sizeof(struct timestamp), */
        /*i * sizeof(struct timestamp_queue), */
        /*hipMemcpyHostToDevice));*/

        index = ctx[i];
        if (index > old_index[i]) {
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[old_index[i]], 
                        (index - old_index[i] + 1) * sizeof(struct timestamp),
                        /*(index - old_index[i]) * sizeof(struct timestamp),*/
                        i * sizeof(struct timestamp_queue) + 
                        old_index[i] * sizeof(struct timestamp), 
                        hipMemcpyHostToDevice));
        } else if (index < old_index[i]) {
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[old_index[i]], 
                        (MAX_TIMESTAMP_NUM - old_index[i] - 1) * sizeof(struct timestamp), 
                        i * sizeof(struct timestamp_queue) + 
                        old_index[i] * sizeof(struct timestamp), 
                        hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[0], 
                        (index - 0 + 1) * sizeof(struct timestamp), 
                        i * sizeof(struct timestamp_queue) + 
                        0,
                        hipMemcpyHostToDevice));
        }
        old_index[i] = index;
    }

    }

#endif

    __host__ void module_cuda_timestamp_entry_fetch_interface(
            uint8_t tid, uint32_t index, struct timestamp *h_ts_entry)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_ts_entry, HIP_SYMBOL(gts), 
                    sizeof(struct timestamp), 
                    tid * sizeof(struct timestamp_queue) + 
                    index * sizeof(struct timestamp), 
                    hipMemcpyDeviceToHost));
    }

    __host__ void module_cuda_global_history_queue_update_interface(
            struct global_history_queue *h_gh_queue)
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gh), h_gh_queue, 
                    sizeof(struct global_history_queue)));
    }

    __host__ void module_cuda_global_history_queue_fetch_interface(
            struct global_history_queue *h_gh_queue)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_gh_queue, HIP_SYMBOL(gh), 
                    sizeof(struct global_history_queue)));
    }

    __host__ void module_cuda_history_hash_queue_update_interface(
            uint8_t tid, struct history_hash_queue *h_hh_queue)
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gh), h_hh_queue, 
                    sizeof(struct history_hash_queue), 
                    tid * sizeof(struct history_hash_queue), 
                    hipMemcpyHostToDevice));
    }

    __host__ void module_cuda_history_hash_queue_fetch_interface(
            uint8_t tid, struct history_hash_queue *h_hh_queue)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_hh_queue, HIP_SYMBOL(gh), 
                    sizeof(struct history_hash_queue), 
                    tid * sizeof(struct history_hash_queue), 
                    hipMemcpyDeviceToHost));
    }

    __host__ void module_cuda_global_page_filter_update_interface(
            struct global_page_filter *h_gpf)
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gpf), h_gpf, 
                    sizeof(struct global_page_filter)));
    }

    __host__ void module_cuda_global_page_filter_fetch_interface(
            struct global_page_filter *h_gpf)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_gpf, HIP_SYMBOL(gpf), 
                    sizeof(struct global_page_filter)));
    }

    __host__ void module_cuda_page_filter_update_interface(
            uint8_t tid, struct page_filter *h_pf)
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gpf), h_pf, 
                    sizeof(struct page_filter), 
                    tid * sizeof(struct page_filter), 
                    hipMemcpyHostToDevice));
    }

    __host__ void module_cuda_page_filter_fetch_interface(
            uint8_t tid, struct page_filter *h_pf)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_pf, HIP_SYMBOL(gpf), 
                    sizeof(struct page_filter), 
                    tid * sizeof(struct page_filter), 
                    hipMemcpyDeviceToHost));
    }

    __host__ void module_cuda_global_race_queue_fetch_interface(
            struct global_race_queue *h_gr)
    {
        cutilSafeCall(hipMemcpyFromSymbol(h_gr, HIP_SYMBOL(gr), 
                    sizeof(struct global_race_queue)));
#if 1
        printf("cuda : race : %d\n", h_gr->sum);
#endif
    }

#ifndef PPI_THREE_STAGE
    __host__ void module_cuda_match_with_trace_buf_interface(
            uint8_t tid, uint32_t size, struct trace_content *h_trace_buf)
    {
        /*printf("cuda : %d, %d, 0x%lx\n", tid, size, h_trace_buf);*/
        cutilSafeCall(hipMemcpy(d_trace_buf, h_trace_buf, 
                    sizeof(struct trace_content) * size, 
                    hipMemcpyHostToDevice));

        numBlocks = (size + numThreads - 1) / numThreads;

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(module_match_with_trace_buf_on_cuda), hipFuncCachePreferL1); 
        module_match_with_trace_buf_on_cuda<<<numBlocks, numThreads>>>(size,
                d_trace_buf);
    }

#else
    __host__ void module_cuda_match_with_trace_buf_interface(
            uint8_t tid, uint32_t size)
    {

#ifdef KERNEL_TIME
        float elapsed = 0.f;
        
        hipEventRecord(start, 0);
#endif

        /*printf("cuda : %d, %d\n", tid, size);*/
        cutilSafeCall(hipMemcpyAsync(d_trace_buf, cuda_buf, 
                    sizeof(struct trace_content) * size, 
                    hipMemcpyHostToDevice, 0));

        numBlocks = (size + numThreads - 1) / numThreads;

        hipFuncSetCacheConfig(reinterpret_cast<const void*>(module_match_with_trace_buf_on_cuda), hipFuncCachePreferL1); 
        module_match_with_trace_buf_on_cuda<<<numBlocks, numThreads, 0, 0>>>(size,
                d_trace_buf);

#ifdef KERNEL_TIME
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        cuda_time += elapsed;
#endif

    }
#endif
