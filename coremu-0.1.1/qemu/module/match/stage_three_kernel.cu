#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Stage Three Kernel 
// include filter / match stage 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

#include "structs.h"

__global__ void stage_three_kernel(uint8_t tid, 
        uint32_t size, struct trace_content *buf)
{
    const int i = threadIdx.x;
    struct trace_content content;

    /*for (i = 0; i < size; i++) {*/
    content = buf[i];

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_match(content);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_match(content);
    } else {
        fprintf(stderr, "unknown type : %d\n", content->type);
        assert(0);
    }
    /*}*/
}

///////////////////////////////////////////////
// Filter Stage Device Functions 

__device__ inline void module_filter_load_match(struct trace_content *content) 
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < info.max_tid_num; i++) {
        if (i != tid) {
            if (pfilter->thread[i]->entry[index].store) {
                module_match_with_store(content, i);
            }
        }
    }
}

__device__ inline void module_filter_store_match(struct trace_content *content) 
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < info.max_tid_num; i++) {
        if (i != tid) {
            if (pfilter->thread[i]->entry[index].load) {
                module_match_with_load(content, i);
            }

            if (pfilter->thread[i]->entry[index].store) {
                module_match_with_store(content, i);
            }
        }
    }
}

///////////////////////////////////////////////
// Match Stage Device Functions 

static inline void module_match_with_load(struct trace_content *content, uint8_t other_tid) 
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(history.thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);

    tail = temp_queue->load_tail;
    head = tail + 1;
    if (head == MAX_LOAD_QUEUE_SIZE) {
        head = 0;
    }

    last_index = ts.thread[other_tid].count;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_LOAD_QUEUE_SIZE;
        }
        tail--;

        temp_entry= &temp_queue->load_entry[tail];

        other_index = temp_entry->content.index;

        if (last_index != other_index) {
            if (module_timestamp_order(other_tid, other_index, tid, index)) {
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            module_race_collection(&temp_entry->content, content);

            break;
        }
    }
}

static inline void module_match_with_store(struct trace_content *content, uint8_t other_tid) 
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(history.thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);

    tail = temp_queue->store_tail;
    head = tail + 1;
    if (head == MAX_STORE_QUEUE_SIZE) {
        head = 0;
    }

    last_index = ts.thread[other_tid].count;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_STORE_QUEUE_SIZE;
        }
        tail--;

        temp_entry = &temp_queue->store_entry[tail];

        other_index = temp_entry->content.index;  

        if (last_index != other_index) {
            if (module_timestamp_order(other_tid, other_index, tid, index)) {   
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            module_race_collection(&temp_entry->content, content);

            break;
        } 
    }
}
