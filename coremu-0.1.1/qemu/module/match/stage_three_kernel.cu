#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Stage Three Kernel 
// include filter / match stage 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

#include "structs.h"

#define NUM_THREADS 32

__device__ struct global_history_queue *ghq;
__device__ struct global_timestamp_queue * gtq;
__device__ struct global_page_filter *pfilter;
__constant__ int d_max_tid_num;


///////////////////////////////////////////////
// Match Stage Device Functions 

__device__ inline int module_timestamp_order(uint8_t tid1, uint32_t index1, 
        uint8_t tid2, uint32_t index2) 
{
    struct timestamp *ts1, *ts2;

    ts1 = &gtq->thread[tid1].entry[index1];
    ts2 = &gtq->thread[tid2].entry[index2];

    if ( ( (ts1->scalar[tid1] < ts2->scalar[tid1])
        && (ts1->scalar[tid2] < ts2->scalar[tid2]) ) || 
         ( (ts1->scalar[tid1] > ts2->scalar[tid1]) 
         &&(ts1->scalar[tid2] > ts2->scalar[tid2]) ) ) {
        return 1;
    }

    return 0;
}

__device__ inline void module_match_with_load(struct trace_content *content, uint8_t other_tid) 
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(ghq->thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);

    tail = temp_queue->load_tail;
    head = tail + 1;
    if (head == MAX_LOAD_QUEUE_SIZE) {
        head = 0;
    }

    last_index = gtq->thread[other_tid].count;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_LOAD_QUEUE_SIZE;
        }
        tail--;

        temp_entry= &temp_queue->load_entry[tail];

        other_index = temp_entry->content.index;

        if (last_index != other_index) {
            if (module_timestamp_order(other_tid, other_index, tid, index)) {
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            /*module_race_collection(&temp_entry->content, content);*/

            break;
        }
    }
}

__device__ inline void module_match_with_store(struct trace_content *content, uint8_t other_tid) 
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index, last_index;
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(ghq->thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);

    tail = temp_queue->store_tail;
    head = tail + 1;
    if (head == MAX_STORE_QUEUE_SIZE) {
        head = 0;
    }

    last_index = gtq->thread[other_tid].count;

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_STORE_QUEUE_SIZE;
        }
        tail--;

        temp_entry = &temp_queue->store_entry[tail];

        other_index = temp_entry->content.index;  

        if (last_index != other_index) {
            if (module_timestamp_order(other_tid, other_index, tid, index)) {   
                break;
            }

            last_index = other_index;
        }

        other_address = temp_entry->content.address;

        if (address == other_address) {
            /*module_race_collection(&temp_entry->content, content);*/

            break;
        } 
    }
}

///////////////////////////////////////////////
// Filter Stage Device Functions 

__device__ inline void module_filter_load_match(struct trace_content *content) 
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (pfilter->thread[i].entry[index].store) {
                module_match_with_store(content, i);
            }
        }
    }
}

__device__ inline void module_filter_store_match(struct trace_content *content) 
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (pfilter->thread[i].entry[index].load) {
                module_match_with_load(content, i);
            }

            if (pfilter->thread[i].entry[index].store) {
                module_match_with_store(content, i);
            }
        }
    }
}


///////////////////////////////////////////////
// Stage Three Kernel 

__global__ void module_cuda_stage_three_kernel( 
        int size,
        struct global_timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct global_page_filter *d_pfilter,
        struct trace_content *buf,
        int *d_result_queue)
{
    struct trace_content *content;
    const int i = blockIdx.x * NUM_THREADS + threadIdx.x;
    if (i >= size)
        return;

    ghq = d_ghq;
    gtq = d_gtq;
    pfilter = d_pfilter;

    /*for (i = 0; i < size; i++) {*/
    content = &buf[i];

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_match(content);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_match(content);
    } else {
        /*fprintf(stderr, "unknown type : %d\n", content->type);*/
        /*assert(0);*/
    }
    /*}*/
}

