#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Stage Three Kernel 
// include filter / match stage 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

#include "structs.h"

#define NUM_THREADS 32

#define SHARED_MEMORY

__constant__ int d_max_tid_num;
__device__ int d_race_counter;

#define cudaAssert(condition) \
    if (!(condition)) { asm("trap;"); }

///////////////////////////////////////////////
// Race Collection Device Functions 

/*__device__ inline void module_race_collection(struct trace_content *content1, struct trace_content *content2, ) */
/*{*/
/*int i;*/
/*struct race_queue *temp_queue;*/

/*if (i >= temp_queue->count) {*/
/*memcpy(&temp_queue->entry[i].content1, content1, sizeof(struct trace_content));*/
/*memcpy(&temp_queue->entry[i].content2, content2, sizeof(struct trace_content));*/
/*temp_queue->entry[i].instance++;*/

/*temp_queue->count++;*/
/*if (temp_queue->count >= MAX_RACE_NUM) {*/
/*fprintf(stderr, "race queue overflow!\n");*/
/*assert(0);*/
/*}*/
/*}*/
/*}*/

///////////////////////////////////////////////
// History Stage Device Functions 

/*__device__ inline void module_history_load_record(struct trace_content *content) */
/*{*/
/*uint8_t tid;*/
/*uint64_t address;*/
/*struct history_queue *temp_queue;*/
/*uint32_t tail;*/
/*struct history_entry *temp_entry;*/

/*tid = content->tid;*/
/*address = content->address;*/

/*temp_queue = &(history.thread[tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);*/

/*tail = temp_queue->load_tail;	*/
/*temp_entry = &temp_queue->load_entry[tail];*/

/*// memcpy(&temp_entry->content, content, sizeof(struct trace_content));*/
/*temp_entry->content.tid = content->tid;*/
/*temp_entry->content.type = content->type;*/
/*temp_entry->content.size = content->size;*/
/*temp_entry->content.address = content->address;*/
/*temp_entry->content.index = content->index;*/
/*temp_entry->content.pc = content->pc;*/

/*//tail++;*/
/*//if (tail >= MAX_LOAD_QUEUE_SIZE) {*/
/*//    tail = 0;*/
/*//}*/
/*//temp_queue->load_tail = tail;*/
/*temp_queue->load_tail = (tail + 1) % MAX_LOAD_QUEUE_SIZE;*/
/*}*/

/*__device__ inline void module_history_store_record(struct trace_content *content) */
/*{*/
/*uint8_t tid;*/
/*uint64_t address;*/
/*struct history_queue *temp_queue;*/
/*uint32_t tail;*/
/*struct history_entry *temp_entry;*/

/*tid = content->tid;*/
/*address = content->address;*/

/*temp_queue = &(d_ghq->thread[tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);*/

/*tail = temp_queue->store_tail;*/
/*temp_entry = &temp_queue->store_entry[tail];*/

/*// memcpy(&temp_entry->content, content, sizeof(struct trace_content));*/
/*temp_entry->content.tid = content->tid;*/
/*temp_entry->content.type = content->type;*/
/*temp_entry->content.size = content->size;*/
/*temp_entry->content.address = content->address;*/
/*temp_entry->content.index = content->index;*/
/*temp_entry->content.pc = content->pc;*/

/*//tail++;*/
/*//if (tail >= MAX_STORE_QUEUE_SIZE) {*/
/*//    tail = 0;*/
/*//}*/
/*//temp_queue->store_tail = tail;*/
/*temp_queue->store_tail = (tail + 1) % MAX_STORE_QUEUE_SIZE;*/
/*}*/

///////////////////////////////////////////////
// Match Stage Device Functions 

__device__ inline int module_timestamp_order(
        struct timestamp_queue *d_gtq,
        uint8_t tid1, uint32_t index1, 
        uint8_t tid2, uint32_t index2) 
{
    struct timestamp *ts1, *ts2;

    ts1 = &d_gtq[tid1].entry[index1];
    ts2 = &d_gtq[tid2].entry[index2];

    return  ( (ts1->scalar[tid1] < ts2->scalar[tid1])
                && (ts1->scalar[tid2] < ts2->scalar[tid2]) ) || 
            ( (ts1->scalar[tid1] > ts2->scalar[tid1]) 
              &&(ts1->scalar[tid2] > ts2->scalar[tid2]) ); 

}

__device__ inline void module_match_with_load(
        struct timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct trace_content *content,
        uint8_t other_tid, 
        struct race_entry *d_result_queue)
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index;
    /*uint32_t last_index;*/
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;
    /*const int i = blockIdx.x * NUM_THREADS + threadIdx.x;*/

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(d_ghq->thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);

    tail = temp_queue->load_tail;
    head = tail + 1;
    if (head == MAX_LOAD_QUEUE_SIZE) {
        head = 0;
    }

    /*last_index = d_gtq[other_tid].count;*/
    /*last_index = 0;*/

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_LOAD_QUEUE_SIZE;
        }
        tail--;

        temp_entry= &temp_queue->load_entry[tail];

        other_index = temp_entry->content.index;

        /*if (last_index != other_index) {*/
            if (module_timestamp_order(d_gtq, other_tid, other_index, tid, index)) {
                break;
            }

            /*last_index = other_index;*/
        /*}*/

        other_address = temp_entry->content.address;

        if (address == other_address) {
            /*module_race_collection(&temp_entry->content, content);*/
            int index = atomicAdd(&d_race_counter, 1);
            /*d_result_queue[index].ts1 = d_gtq[tid].entry[temp_entry->content.index];*/
            /*d_result_queue[index].ts2 = d_gtq[tid].entry[content->index];*/
            d_result_queue[index].pc1 = temp_entry->content.pc;
            d_result_queue[index].pc2 = content->pc;
            d_result_queue[index].address = address;
            break;
        }
    }
}

__device__ inline void module_match_with_store(
        struct timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct trace_content *content,
        uint8_t other_tid, 
        struct race_entry *d_result_queue)
{
    uint8_t tid;
    uint64_t address, other_address;
    uint32_t index, other_index;
    /*uint32_t last_index;*/
    struct history_queue *temp_queue;
    uint32_t head, tail;
    struct history_entry *temp_entry;
    /*const int i = blockIdx.x * NUM_THREADS + threadIdx.x;*/

    tid = content->tid;
    address = content->address;
    index = content->index;

    temp_queue = &(d_ghq->thread[other_tid].hash[(address >> HASH_BASE_BIT) % MAX_HASH_NUM]);
    cudaAssert((address >> HASH_BASE_BIT) % MAX_HASH_NUM < MAX_HASH_NUM);
    /*temp_queue = &(d_ghq->thread[other_tid].hash[0]);*/

    tail = temp_queue->store_tail;
    head = tail + 1;
    if (head == MAX_STORE_QUEUE_SIZE) {
        head = 0;
    }

    /*last_index = d_gtq[other_tid].count;*/
    /*last_index = 0;*/

    while (tail != head) {
        if (tail == 0) {
            tail = MAX_STORE_QUEUE_SIZE;
        }
        tail--;

        temp_entry = &temp_queue->store_entry[tail];

        other_index = temp_entry->content.index;  

        /*if (last_index != other_index) {*/
            if (module_timestamp_order(d_gtq, other_tid, other_index, tid, index)) {
                break;
            }

            /*last_index = other_index;*/
        /*}*/

        other_address = temp_entry->content.address;

        if (address == other_address) {
            /*module_race_collection(&temp_entry->content, content);*/
            /*d_result_queue[i] = 1;*/
            int index = atomicAdd(&d_race_counter, 1);
            /*d_result_queue[index].ts1 = d_gtq[tid].entry[temp_entry->content.index];*/
            /*d_result_queue[index].ts2 = d_gtq[tid].entry[content->index];*/
            d_result_queue[index].pc1 = temp_entry->content.pc;
            d_result_queue[index].pc2 = content->pc;
            d_result_queue[index].address = address;
            break;
        } 
    }
}

///////////////////////////////////////////////
// Filter Stage Device Functions 

__device__ inline void module_filter_load_match(
        struct timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct global_page_filter *d_pfilter,
        struct trace_content *content, 
        struct race_entry *d_result_queue)
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (d_pfilter->thread[i].entry[index].store) {
                module_match_with_store(d_gtq, d_ghq, content, i, d_result_queue);
            }
        }
    }
}

__device__ inline void module_filter_store_match(
        struct timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct global_page_filter *d_pfilter,
        struct trace_content *content, 
        struct race_entry *d_result_queue)
{
    uint8_t i;
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    for (i = 0; i < d_max_tid_num; i++) {
        if (i != tid) {
            if (d_pfilter->thread[i].entry[index].load) {
                module_match_with_load(d_gtq, d_ghq, content, i, d_result_queue);
            }

            if (d_pfilter->thread[i].entry[index].store) {
                module_match_with_store(d_gtq, d_ghq, content, i, d_result_queue);
            }
        }
    }
}


///////////////////////////////////////////////
// Stage Three Kernel 

__global__ void module_cuda_stage_three_kernel( 
        int size,
        struct timestamp_queue *d_gtq,
        struct global_history_queue *d_ghq,
        struct global_page_filter *d_pfilter,
        struct trace_content *buf,
        struct race_entry *d_result_queue)
{
#ifdef SHARED_MEMORY
    __shared__ struct trace_content content [NUM_THREADS];
#else
    struct trace_content *content;
#endif
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    /*int i = threadIdx.x;*/
    if (i >= size)
        return;

#ifdef SHARED_MEMORY

    content[tid] = buf[i];

    if (content[tid].type == TRACE_MEM_LOAD) {
        module_filter_load_match(d_gtq, d_ghq, d_pfilter, &content[tid],
                d_result_queue);
    } else if (content[tid].type == TRACE_MEM_STORE) {
        module_filter_store_match(d_gtq, d_ghq, d_pfilter, &content[tid],
                d_result_queue);
    } else {
    }
#else
    content = &buf[i];

    if (content->type == TRACE_MEM_LOAD) {
        module_filter_load_match(d_gtq, d_ghq, d_pfilter, content,
                d_result_queue);
    } else if (content->type == TRACE_MEM_STORE) {
        module_filter_store_match(d_gtq, d_ghq, d_pfilter, content,
                d_result_queue);
    } else {
    }
#endif

}

