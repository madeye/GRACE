////////////////////////////////////////////////////////////////////////////////
// Stage Three CUDA Interface 
// include timestamp / history / filter 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////
// Included Headers 

#include <stdint.h>


#include <cutil_inline.h>

#include "../interface.h"
#include "../info.c"
#include "stage_three_kernel.cu"

///////////////////////////////////////////////
// Test Functions 

//#define COMPILE_TO_BINARY
#ifdef COMPILE_TO_BINARY
__host__ static void tool_global_timestamp_queue_init(
        struct global_timestamp_queue *gts_queue)
{
    gts_queue->thread[0].entry[0].scalar[0] = 0;
    gts_queue->thread[0].entry[0].scalar[1] = 0;
    gts_queue->thread[0].entry[1].scalar[0] = 1;
    gts_queue->thread[0].entry[1].scalar[1] = 1;
    gts_queue->thread[0].entry[2].scalar[0] = 2;
    gts_queue->thread[0].entry[2].scalar[1] = 1;
    gts_queue->thread[0].entry[3].scalar[0] = 3;
    gts_queue->thread[0].entry[3].scalar[1] = 3;
    gts_queue->thread[0].entry[4].scalar[0] = 4;
    gts_queue->thread[0].entry[4].scalar[1] = 4;

    gts_queue->thread[1].entry[0].scalar[0] = 0;
    gts_queue->thread[1].entry[0].scalar[1] = 0;
    gts_queue->thread[1].entry[1].scalar[0] = 0;
    gts_queue->thread[1].entry[1].scalar[1] = 1;
    gts_queue->thread[1].entry[2].scalar[0] = 2;
    gts_queue->thread[1].entry[2].scalar[1] = 2;
    gts_queue->thread[1].entry[3].scalar[0] = 3;
    gts_queue->thread[1].entry[3].scalar[1] = 3;
    gts_queue->thread[1].entry[4].scalar[0] = 4;
    gts_queue->thread[1].entry[4].scalar[1] = 4;
}

#define STRIDE 4

__host__ static void tool_history_queue_init(
        struct history_entry *history_queue, int size)
{
    int i;

    printf("\nHistory Queue Entry :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        history_queue[i].content.address = (i % STRIDE) << 2;
        history_queue[i].content.tid = 1;
        history_queue[i].content.type = 2;
        history_queue[i].content.size = 4;
        history_queue[i].content.index = (i / STRIDE);
        history_queue[i].content.pc = 0xf;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                history_queue[i].content.address, history_queue[i].content.tid,
                history_queue[i].content.type, history_queue[i].content.size,
                history_queue[i].content.index, history_queue[i].content.pc);
    }
}

__host__ static void tool_trace_buf_init(
        struct trace_content *trace_buf, int size)
{
    int i;

    printf("\nTrace Queue Content :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        trace_buf[i].address = (STRIDE >> 1) << 2;
        trace_buf[i].tid = 0;
        trace_buf[i].type = 1;
        trace_buf[i].size = 4;
        trace_buf[i].index = 1;
        trace_buf[i].pc = 0x4;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                trace_buf[i].address, trace_buf[i].tid, 
                trace_buf[i].type, trace_buf[i].size, 
                trace_buf[i].index, trace_buf[i].pc);
    }
}

__host__ static void tool_result_queue_print(
        int *result_queue, int size)
{
    int i;

    printf("\nResult Queue Entry :\n");
    printf("No\tResult\n");

    for (i = 0; i < size; i++)
    {
        printf("%d\t%d\n", i, result_queue[i]);
    }
}
#endif

///////////////////////////////////////////////
// Global Device Data Structure 

struct trace_content *d_buf;
struct global_timestamp_queue *d_timestamp;  
struct global_history_queue *d_history;

extern "C" void module_detector_stage_three_cuda(uint8_t tid, 
        uint32_t size, struct trace_content *buf);

void module_detector_stage_three_cuda(uint8_t tid, 
        uint32_t size, struct trace_content *buf)
{
    uint32_t i;

    CUDA_SAFE_CALL(hipMalloc((void **) &d_buf,
                size * sizeof(struct trace_content)));
    CUDA_SAFE_CALL(hipMemcpy(d_buf, buf,
                size * sizeof(struct trace_content),
                hipMemcpyHostToDevice))
    
    for (i = 0; i < size; i++) {
        content = &buf[i];

        if (content->type == TRACE_MEM_LOAD) {
            module_filter_load_match(content);
        } else if (content->type == TRACE_MEM_STORE) {
            module_filter_store_match(content);
        } else {
            fprintf(stderr, "unknown type : %d\n", content->type);
            assert(0);
        }
    }
}

extern "C" void module_detector_init_cuda(); 

void module_detector_init_cuda() {

    cutilSafeCall(hipMalloc((void **)&d_history_queue, 
                sizeof(struct global_history_queue)));
    cutilSafeCall(hipMalloc((void **)&d_trace_buf, 
                TRACE_BUF_SIZE * sizeof(struct trace_content)));
    /*cutilSafeCall(hipMalloc((void **)&d_result_queue,*/
                /*sizeof(int) * HISTORY_QUEUE_SIZE));*/

    hipSetDevice(cutGetMaxGflopsDeviceId());

    printf("\nglobal timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));

}

extern "C" void module_detector_update_cuda(uint8_t tid, 
        uint32_t size, struct trace_content *buf);

void module_detector_update_cuda(
        struct global_history_queue *h_hq 
        struct global_timestamp_queue *h_tq) {


}
