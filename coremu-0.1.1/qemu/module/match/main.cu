////////////////////////////////////////////////////////////////////////////////
// Stage Three CUDA Interface 
// include timestamp / history / filter 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////
// Included Headers 

#include <stdint.h>
#include <cutil_inline.h>

#include "../interface.h"
#include "stage_three_kernel.cu"

///////////////////////////////////////////////
// Test Functions 

#define COMPILE_TO_BINARY
#ifdef COMPILE_TO_BINARY

static void module_filter_store_record(
        struct global_page_filter *h_pfilter,
        struct trace_content *content) 
{
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    h_pfilter->thread[tid].entry[index].store = 1;
}

static void module_filter_load_record(
        struct global_page_filter *h_pfilter,
        struct trace_content *content) 
{
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    h_pfilter->thread[tid].entry[index].load = 1;
}

static void tool_global_page_filter_init(
        struct global_page_filter *h_pfilter,
        struct global_history_queue *h_ghq, int size)
{
    int t, h, i;

    for (t = 0; t < 2; t++)
        for (h = 0; h < MAX_HASH_NUM; h++)
            for (i = 0; i < size; i++)
            {
                module_filter_load_record(h_pfilter,
                        &h_ghq->thread[t].hash[h].load_entry[i].content);
            }

}

__host__ static void tool_global_timestamp_queue_init(
        struct global_timestamp_queue *gts_queue)
{
    gts_queue->thread[0].entry[0].scalar[0] = 0;
    gts_queue->thread[0].entry[0].scalar[1] = 0;
    gts_queue->thread[0].entry[1].scalar[0] = 1;
    gts_queue->thread[0].entry[1].scalar[1] = 1;
    gts_queue->thread[0].entry[2].scalar[0] = 2;
    gts_queue->thread[0].entry[2].scalar[1] = 1;
    gts_queue->thread[0].entry[3].scalar[0] = 3;
    gts_queue->thread[0].entry[3].scalar[1] = 3;
    gts_queue->thread[0].entry[4].scalar[0] = 4;
    gts_queue->thread[0].entry[4].scalar[1] = 4;

    gts_queue->thread[1].entry[0].scalar[0] = 0;
    gts_queue->thread[1].entry[0].scalar[1] = 0;
    gts_queue->thread[1].entry[1].scalar[0] = 0;
    gts_queue->thread[1].entry[1].scalar[1] = 1;
    gts_queue->thread[1].entry[2].scalar[0] = 2;
    gts_queue->thread[1].entry[2].scalar[1] = 2;
    gts_queue->thread[1].entry[3].scalar[0] = 3;
    gts_queue->thread[1].entry[3].scalar[1] = 3;
    gts_queue->thread[1].entry[4].scalar[0] = 4;
    gts_queue->thread[1].entry[4].scalar[1] = 4;
}

#define STRIDE 4

__host__ static void tool_global_history_queue_init(
        struct global_history_queue *h_ghq, int size)
{
    int t, h, i;

    printf("\nHistory Queue Entry :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (t = 0; t < 2; t++)
        for (h = 0; h < MAX_HASH_NUM; h++)
            for (i = 0; i < size; i++)
            {
                h_ghq->thread[t].hash[h].load_entry[i].content.address = (i % STRIDE) << 2;
                h_ghq->thread[t].hash[h].load_entry[i].content.tid = 1;
                h_ghq->thread[t].hash[h].load_entry[i].content.type = 2;
                h_ghq->thread[t].hash[h].load_entry[i].content.size = 4;
                h_ghq->thread[t].hash[h].load_entry[i].content.index = (i / STRIDE);
                h_ghq->thread[t].hash[h].load_entry[i].content.pc = 0xf;

                printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                        h_ghq->thread[t].hash[h].load_entry[i].content.address, 
                        h_ghq->thread[t].hash[h].load_entry[i].content.tid,
                        h_ghq->thread[t].hash[h].load_entry[i].content.type, 
                        h_ghq->thread[t].hash[h].load_entry[i].content.size,
                        h_ghq->thread[t].hash[h].load_entry[i].content.index, 
                        h_ghq->thread[t].hash[h].load_entry[i].content.pc);
            }
}

__host__ static void tool_trace_buf_init(
        struct trace_content *trace_buf, int size)
{
    int i;

    printf("\nTrace Queue Content :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        trace_buf[i].address = (STRIDE >> 1) << 2;
        trace_buf[i].tid = 0;
        trace_buf[i].type = 1;
        trace_buf[i].size = 4;
        trace_buf[i].index = 1;
        trace_buf[i].pc = 0x4;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                trace_buf[i].address, trace_buf[i].tid, 
                trace_buf[i].type, trace_buf[i].size, 
                trace_buf[i].index, trace_buf[i].pc);
    }
}

__host__ static void tool_result_queue_print(
        int *result_queue, int size)
{
    int i;

    printf("\nResult Queue Entry :\n");
    printf("No\tResult\n");

    for (i = 0; i < size; i++)
    {
        printf("%d\t%d\n", i, result_queue[i]);
    }
}
#endif

///////////////////////////////////////////////
// Global Device Data Structure 

struct trace_content *d_trace_buf;
struct global_timestamp_queue *d_gtq;  
struct global_history_queue *d_ghq;
struct global_page_filter *d_pfilter;

extern "C" void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf, int *d_result_queue);

void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf, int *d_result_queue)
{
    CUDA_SAFE_CALL(hipMemcpy(d_trace_buf, buf,
                size * sizeof(struct trace_content),
                hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_max_tid_num), &h_max_tid_num, 
                sizeof(int)));

    module_cuda_stage_three_kernel
        <<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>
        (size, d_gtq, d_ghq, d_pfilter, d_trace_buf, d_result_queue);
    
}

extern "C" void module_cuda_free(); 

void module_cuda_free() {

    CUDA_SAFE_CALL(hipFree(d_gtq));
    CUDA_SAFE_CALL(hipFree(d_ghq));
    CUDA_SAFE_CALL(hipFree(d_trace_buf));
    CUDA_SAFE_CALL(hipFree(d_pfilter));
    /*CUDA_SAFE_CALL(hipMalloc((void **)&d_result_queue,*/
                /*sizeof(int) * HISTORY_QUEUE_SIZE));*/

}

extern "C" void module_cuda_init(); 

void module_cuda_init() {

    CUDA_SAFE_CALL(hipMalloc((void **)&d_gtq, 
                sizeof(struct global_timestamp_queue)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_ghq, 
                sizeof(struct global_history_queue)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_trace_buf, 
                TRACE_BUF_SIZE * sizeof(struct trace_content)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_pfilter, 
                sizeof(struct global_page_filter)));
    /*CUDA_SAFE_CALL(hipMalloc((void **)&d_result_queue,*/
                /*sizeof(int) * HISTORY_QUEUE_SIZE));*/

    hipSetDevice(cutGetMaxGflopsDeviceId());

    printf("\nglobal timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));

}

extern "C" void module_cuda_update(
        struct global_history_queue *h_ghq, 
        struct global_timestamp_queue *h_gtq, 
        struct global_page_filter *h_pfilter); 

void module_cuda_update(
        struct global_history_queue *h_ghq, 
        struct global_timestamp_queue *h_gtq, 
        struct global_page_filter *h_pfilter) { 

    CUDA_SAFE_CALL(hipMemcpy(d_ghq, h_ghq,
                sizeof(struct global_history_queue),
                hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpy(d_gtq, h_gtq,
                sizeof(struct global_timestamp_queue),
                hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpy(d_pfilter, h_pfilter,
                sizeof(struct global_page_filter),
                hipMemcpyHostToDevice));

}

#ifdef COMPILE_TO_BINARY
int main(int argc, char** argv)  
{
    struct global_timestamp_queue *h_gtq;
    struct global_history_queue *h_ghq;
    struct global_page_filter *h_pfilter;
    struct trace_content *h_trace_buf;
    int *h_result_queue;

    h_gtq = (struct global_timestamp_queue *)malloc(
            sizeof(struct global_timestamp_queue));
    memset(h_gtq, 0, sizeof(struct global_timestamp_queue));
    tool_global_timestamp_queue_init(h_gtq);

    h_ghq = (struct global_history_queue *)malloc(
            sizeof(struct global_history_queue));
    memset(h_ghq, 0, 
            sizeof(struct global_history_queue));
    tool_global_history_queue_init(h_ghq, MAX_LOAD_QUEUE_SIZE);

    h_pfilter = (struct global_page_filter *)malloc(
            sizeof(struct global_page_filter));
    memset(h_pfilter, 0, 
            sizeof(struct global_page_filter));
    tool_global_page_filter_init(h_pfilter, h_ghq, MAX_LOAD_QUEUE_SIZE);

    h_trace_buf = (struct trace_content *)malloc(
            TRACE_BUF_SIZE * sizeof(struct trace_content));
    memset(h_trace_buf, 0, TRACE_BUF_SIZE * sizeof(struct trace_content));
    tool_trace_buf_init(h_trace_buf, TRACE_BUF_SIZE);

    h_result_queue = (int *)malloc(sizeof(int) * TRACE_BUF_SIZE);
    memset(h_result_queue, 0, sizeof(int) * TRACE_BUF_SIZE);

    module_cuda_init();
    module_cuda_update(h_ghq, h_gtq, h_pfilter);
    module_cuda_stage_three(0, TRACE_BUF_SIZE, h_trace_buf, h_result_queue);
    module_cuda_free();

    tool_result_queue_print(h_result_queue, TRACE_BUF_SIZE);

    free(h_gtq);
    free(h_ghq);
    free(h_pfilter);
    free(h_trace_buf);
    free(h_result_queue);

    hipDeviceReset();
    //cutilExit(argc, argv);
}
#endif

