////////////////////////////////////////////////////////////////////////////////
// Stage Three CUDA Interface 
// include timestamp / history / filter 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////
// Included Headers 

#include <stdint.h>
#include <cutil_inline.h>

#include "../interface.h"
#include "../info.c"
#include "stage_three_kernel.cu"

///////////////////////////////////////////////
// Test Functions 

//#define COMPILE_TO_BINARY
#ifdef COMPILE_TO_BINARY
__host__ static void tool_global_timestamp_queue_init(
        struct global_timestamp_queue *gts_queue)
{
    gts_queue->thread[0].entry[0].scalar[0] = 0;
    gts_queue->thread[0].entry[0].scalar[1] = 0;
    gts_queue->thread[0].entry[1].scalar[0] = 1;
    gts_queue->thread[0].entry[1].scalar[1] = 1;
    gts_queue->thread[0].entry[2].scalar[0] = 2;
    gts_queue->thread[0].entry[2].scalar[1] = 1;
    gts_queue->thread[0].entry[3].scalar[0] = 3;
    gts_queue->thread[0].entry[3].scalar[1] = 3;
    gts_queue->thread[0].entry[4].scalar[0] = 4;
    gts_queue->thread[0].entry[4].scalar[1] = 4;

    gts_queue->thread[1].entry[0].scalar[0] = 0;
    gts_queue->thread[1].entry[0].scalar[1] = 0;
    gts_queue->thread[1].entry[1].scalar[0] = 0;
    gts_queue->thread[1].entry[1].scalar[1] = 1;
    gts_queue->thread[1].entry[2].scalar[0] = 2;
    gts_queue->thread[1].entry[2].scalar[1] = 2;
    gts_queue->thread[1].entry[3].scalar[0] = 3;
    gts_queue->thread[1].entry[3].scalar[1] = 3;
    gts_queue->thread[1].entry[4].scalar[0] = 4;
    gts_queue->thread[1].entry[4].scalar[1] = 4;
}

#define STRIDE 4

__host__ static void tool_history_queue_init(
        struct global_history_queue *h_ghq, int size)
{
    int t, h, i;

    printf("\nHistory Queue Entry :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (t = 0; t < 2; t++)
        for (h = 0; h < MAX_HASH_NUM; h++)
            for (i = 0; i < size; i++)
            {
                h_ghq->thread[t].hash[h].load_entry[i].content.address = (i % STRIDE) << 2;
                h_ghq->thread[t].hash[h].load_entry[i].content.tid = 1;
                h_ghq->thread[t].hash[h].load_entry[i].content.type = 2;
                h_ghq->thread[t].hash[h].load_entry[i].content.size = 4;
                h_ghq->thread[t].hash[h].load_entry[i].content.index = (i / STRIDE);
                h_ghq->thread[t].hash[h].load_entry[i].content.pc = 0xf;

                printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                        history_queue[i].content.address, history_queue[i].content.tid,
                        history_queue[i].content.type, history_queue[i].content.size,
                        history_queue[i].content.index, history_queue[i].content.pc);
            }
}

__host__ static void tool_trace_buf_init(
        struct trace_content *trace_buf, int size)
{
    int i;

    printf("\nTrace Queue Content :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        trace_buf[i].address = (STRIDE >> 1) << 2;
        trace_buf[i].tid = 0;
        trace_buf[i].type = 1;
        trace_buf[i].size = 4;
        trace_buf[i].index = 1;
        trace_buf[i].pc = 0x4;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                trace_buf[i].address, trace_buf[i].tid, 
                trace_buf[i].type, trace_buf[i].size, 
                trace_buf[i].index, trace_buf[i].pc);
    }
}

__host__ static void tool_result_queue_print(
        int *result_queue, int size)
{
    int i;

    printf("\nResult Queue Entry :\n");
    printf("No\tResult\n");

    for (i = 0; i < size; i++)
    {
        printf("%d\t%d\n", i, result_queue[i]);
    }
}
#endif

///////////////////////////////////////////////
// Global Device Data Structure 

struct trace_content *d_trace_buf;
struct global_timestamp_queue *d_gtq;  
struct global_history_queue *d_ghq;

extern "C" void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf, int *result_queue);

void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf, int *result_queue)
{
    uint32_t i;

    CUDA_SAFE_CALL(hipMemcpy(d_trace_buf, buf,
                size * sizeof(struct trace_content),
                hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_max_tid_num), h_max_tid_num, 
                sizeof(int)));

    module_cuda_stage_three_kernel
        <<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>
        (size, d_gtq, d_ghq, d_trace_buf, d_result_queue);
    
}

extern "C" void module_cuda_free(); 

void module_cuda_free() {

    cutilSafeCall(hipFree(d_gtq));
    cutilSafeCall(hipFree(d_ghq));
    cutilSafeCall(hipMalloc(d_trace_buf));
    /*cutilSafeCall(hipMalloc((void **)&d_result_queue,*/
                /*sizeof(int) * HISTORY_QUEUE_SIZE));*/

}

extern "C" void module_cuda_init(); 

void module_cuda_init() {

    cutilSafeCall(hipMalloc((void **)&d_gtq, 
                sizeof(struct global_timestamp_queue)));
    cutilSafeCall(hipMalloc((void **)&d_ghq, 
                sizeof(struct global_history_queue)));
    cutilSafeCall(hipMalloc((void **)&d_trace_buf, 
                TRACE_BUF_SIZE * sizeof(struct trace_content)));
    /*cutilSafeCall(hipMalloc((void **)&d_result_queue,*/
                /*sizeof(int) * HISTORY_QUEUE_SIZE));*/

    hipSetDevice(cutGetMaxGflopsDeviceId());

    printf("\nglobal timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));

}

extern "C" void module_cuda_update(
        struct global_history_queue *h_ghq 
        struct global_timestamp_queue *h_gtq); 

void module_cuda_update(
        struct global_history_queue *h_ghq 
        struct global_timestamp_queue *h_gtq) {

    CUDA_SAFE_CALL(hipMemcpy(d_ghq, h_ghq,
                sizeof(struct global_history_queue),
                hipMemcpyHostToDevice))

    CUDA_SAFE_CALL(hipMemcpy(d_gtq, h_gtq,
                sizeof(struct global_timestamp_queue),
                hipMemcpyHostToDevice))

}

#ifdef COMPILE_TO_BINARY
int main(int argc, char** argv)  
{
    struct global_timestamp_queue *h_gtq;
    struct global_history_queue *h_ghq;
    struct trace_content *h_trace_buf;
    int *h_result_queue;

    h_gtq = (struct global_timestamp_queue *)malloc(
            sizeof(struct global_timestamp_queue));
    memset(h_gts_queue, 0, sizeof(struct global_timestamp_queue));
    tool_global_timestamp_queue_init(h_gtq);

    h_ghq = (struct history_entry *)malloc(
            sizeof(struct global_history_queue));
    memset(h_history_queue, 0, 
            sizeof(struct global_history_queue));
    tool_history_queue_init(h_ghq, sizeof(struct global_history_queue));

    h_trace_buf = (struct trace_content *)malloc(
            TRACE_BUF_SIZE * sizeof(struct trace_content));
    memset(h_trace_buf, 0, TRACE_BUF_SIZE * sizeof(struct trace_content));
    tool_trace_buf_init(h_trace_buf, TRACE_BUF_SIZE);

    h_result_queue = (int *)malloc(sizeof(int) * TRACE_BUF_SIZE);
    memset(h_result_queue, 0, sizeof(int) * TRACE_BUF_SIZE);

    module_cuda_init();
    module_cuda_update(h_ghq, h_gtq);
    module_cuda_stage_three(0, TRACE_BUF_SIZE, h_trace_buf, h_result_queue);
    module_cuda_free();

    tool_result_queue_print(h_result_queue, TRACE_BUF_SIZE);

    free(h_history_queue);
    free(h_trace_buf);
    free(h_result_queue);

    hipDeviceReset();
    //cutilExit(argc, argv);
}
#endif

