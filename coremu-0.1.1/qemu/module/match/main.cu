////////////////////////////////////////////////////////////////////////////////
// Stage Three CUDA Interface 
// include timestamp / history / filter 
// Author: Max
// Revision: 0.1
////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////
// Included Headers 

#include <stdint.h>
#include <cutil_inline.h>

#include "../interface.h"
#include "stage_three_kernel.cu"

///////////////////////////////////////////////
// Test Functions 

/*#define COMPILE_TO_BINARY*/
#ifdef COMPILE_TO_BINARY

static void module_filter_store_record(
        struct global_page_filter *pfilter,
        struct trace_content *content) 
{
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    pfilter->thread[tid].entry[index].store = 1;
}

static void module_filter_load_record(
        struct global_page_filter *pfilter,
        struct trace_content *content) 
{
    uint8_t tid;
    uint64_t address;
    uint32_t index;

    tid = content->tid;
    address = content->address;

    index = (address >> FILTER_BASE_BIT) & FILTER_ENTRY_MASK;

    pfilter->thread[tid].entry[index].load = 1;
}

static void tool_global_page_filter_init(
        struct global_page_filter *pfilter,
        struct global_history_queue *history, int size)
{
    int t, h, i;

    /*for (t = 0; t < 2; t++)*/
        /*for (h = 0; h < MAX_HASH_NUM; h++)*/
            /*for (i = 0; i < size; i++)*/
            /*{*/
                /*module_filter_load_record(pfilter,*/
                        /*&history->thread[t].hash[h].load_entry[i].content);*/
            /*}*/
    for (t = 0; t < 2; t++)
        for (h = 0; h < MAX_HASH_NUM; h++)
        {
            pfilter->thread[t].entry[h].load = 1;
            printf("thread: %d, entry: %d, status: %d \n", 
                    t, h, pfilter->thread[t].entry[h].load);
        }
    

}

__host__ static void tool_global_timestamp_queue_init(
        struct timestamp_queue *gts_queue)
{
    gts_queue[0].entry[0].scalar[0] = 0;
    gts_queue[0].entry[0].scalar[1] = 0;
    gts_queue[0].entry[1].scalar[0] = 1;
    gts_queue[0].entry[1].scalar[1] = 1;
    gts_queue[0].entry[2].scalar[0] = 2;
    gts_queue[0].entry[2].scalar[1] = 1;
    gts_queue[0].entry[3].scalar[0] = 3;
    gts_queue[0].entry[3].scalar[1] = 3;
    gts_queue[0].entry[4].scalar[0] = 4;
    gts_queue[0].entry[4].scalar[1] = 4;

    gts_queue[1].entry[0].scalar[0] = 0;
    gts_queue[1].entry[0].scalar[1] = 0;
    gts_queue[1].entry[1].scalar[0] = 0;
    gts_queue[1].entry[1].scalar[1] = 1;
    gts_queue[1].entry[2].scalar[0] = 2;
    gts_queue[1].entry[2].scalar[1] = 2;
    gts_queue[1].entry[3].scalar[0] = 3;
    gts_queue[1].entry[3].scalar[1] = 3;
    gts_queue[1].entry[4].scalar[0] = 4;
    gts_queue[1].entry[4].scalar[1] = 4;
}

#define STRIDE 4

__host__ static void tool_global_history_queue_init(
        struct global_history_queue *history, int size)
{
    int t, h, i;

    printf("\nHistory Queue Entry :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (t = 0; t < 2; t++)
        for (h = 0; h < MAX_HASH_NUM; h++)
            for (i = 0; i < size; i++)
            {
                history->thread[t].hash[h].load_entry[i].content.address = (i % STRIDE) << 2;
                history->thread[t].hash[h].load_entry[i].content.tid = 1;
                history->thread[t].hash[h].load_entry[i].content.type = 1;
                history->thread[t].hash[h].load_entry[i].content.size = 4;
                history->thread[t].hash[h].load_entry[i].content.index = (i / STRIDE);
                history->thread[t].hash[h].load_entry[i].content.pc = 0xf;

                printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                        history->thread[t].hash[h].load_entry[i].content.address, 
                        history->thread[t].hash[h].load_entry[i].content.tid,
                        history->thread[t].hash[h].load_entry[i].content.type, 
                        history->thread[t].hash[h].load_entry[i].content.size,
                        history->thread[t].hash[h].load_entry[i].content.index, 
                        history->thread[t].hash[h].load_entry[i].content.pc);
            }
}

__host__ static void tool_trace_buf_init(
        struct trace_content *trace_buf, int size)
{
    int i;

    printf("\nTrace Queue Content :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        trace_buf[i].address = (STRIDE >> 1) << 2;
        trace_buf[i].tid = 0;
        trace_buf[i].type = 2;
        trace_buf[i].size = 4;
        trace_buf[i].index = 1;
        trace_buf[i].pc = 0x4;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                trace_buf[i].address, trace_buf[i].tid, 
                trace_buf[i].type, trace_buf[i].size, 
                trace_buf[i].index, trace_buf[i].pc);
    }
}

__host__ static void tool_result_queue_print(
        int *result_queue, int size)
{
    int i;

    printf("\nResult Queue Entry :\n");
    printf("No\tResult\n");

    for (i = 0; i < size; i++)
    {
        printf("%d\t%d\n", i, result_queue[i]);
    }
}
#endif

///////////////////////////////////////////////
// Global Device Data Structure 

struct trace_content *d_trace_buf;
struct timestamp_queue *d_gtq;  
struct global_history_queue *d_ghq;
struct global_page_filter *d_pfilter;
struct global_race *d_result_queue;

struct global_history_queue *history; 
struct timestamp_queue *h_gtq; 
struct global_page_filter *pfilter;

int h_race_counter;

extern "C" void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf);

void module_cuda_stage_three(int h_max_tid_num, 
        uint32_t size, struct trace_content *buf)
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(module_cuda_stage_three_kernel), hipFuncCachePreferL1); 

    CUDA_SAFE_CALL(hipMemcpy(d_trace_buf, buf,
                size * sizeof(struct trace_content),
                hipMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_max_tid_num), &h_max_tid_num, 
                sizeof(int)));

    module_cuda_stage_three_kernel
        <<<(size + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>
        (size, d_gtq, d_ghq, d_pfilter, d_trace_buf, d_result_queue);

    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
}

extern "C" void module_cuda_free(
        /*struct global_history_queue *history, */
        /*[>struct timestamp_queue *h_gtq, <]*/
        /*struct global_page_filter *pfilter */
        ); 

void module_cuda_free(
        /*struct global_history_queue *history, */
        /*[>struct timestamp_queue *h_gtq, <]*/
        /*struct global_page_filter *pfilter */
        ) {

    CUDA_SAFE_CALL(hipMemcpyFromSymbol(&h_race_counter, HIP_SYMBOL(d_race_counter), sizeof(int)));
    printf("Dynamic Race Num: %d\n", h_race_counter);
    
    CUDA_SAFE_CALL(hipHostFree(h_gtq));
    CUDA_SAFE_CALL(hipHostFree(history));
    CUDA_SAFE_CALL(hipHostFree(pfilter));

    /*CUDA_SAFE_CALL(hipFree(d_gtq));*/
    /*CUDA_SAFE_CALL(hipFree(d_ghq));*/
    CUDA_SAFE_CALL(hipFree(d_trace_buf));
    /*CUDA_SAFE_CALL(hipFree(d_pfilter));*/
    CUDA_SAFE_CALL(hipFree(d_result_queue));

}

extern "C" void module_cuda_init(
        /*struct global_history_queue *ghq, */
        /*struct timestamp_queue *gtq, */
        /*struct global_page_filter *pfilter */
        ); 

void module_cuda_init(
        /*struct global_history_queue *ghq, */
        /*struct timestamp_queue *gtq, */
        /*struct global_page_filter *pfilter */
        ) {

    /*hipSetDevice(cutGetMaxGflopsDeviceId());*/
    hipSetDeviceFlags( hipDeviceMapHost );

    // Allocate for mapped memory
    CUDA_SAFE_CALL(hipHostAlloc((void **)&h_gtq, sizeof(struct
                    timestamp_queue) * MAX_PROCESS_NUM, hipHostMallocMapped));
    CUDA_SAFE_CALL(hipHostAlloc((void **)&history, sizeof(struct
                    global_history_queue), hipHostMallocMapped));
    CUDA_SAFE_CALL(hipHostAlloc((void **)&pfilter, sizeof(struct
                    global_page_filter), hipHostMallocMapped));

    // Return the host pointer
    /*gtq = h_gtq;*/
    /*ghq = history;*/
    /*pfilter = pfilter;*/

    // Init mapped memory
    memset(h_gtq, 0, sizeof(struct timestamp_queue) * MAX_PROCESS_NUM);
    memset(history, 0, sizeof(struct global_history_queue));
    memset(pfilter, 0, sizeof(struct global_page_filter));

    CUDA_SAFE_CALL(hipHostGetDevicePointer((void **)&d_gtq,
                (void *)h_gtq, 0));
    CUDA_SAFE_CALL(hipHostGetDevicePointer((void **)&d_ghq,
                (void *)history, 0));
    CUDA_SAFE_CALL(hipHostGetDevicePointer((void **)&d_pfilter,
                (void *)pfilter, 0));

    /*CUDA_SAFE_CALL(hipMalloc((void **)&d_gtq, */
                /*MAX_PROCESS_NUM * sizeof(struct timestamp_queue)));*/
    /*CUDA_SAFE_CALL(hipMalloc((void **)&d_ghq, */
                /*sizeof(struct global_history_queue)));*/
    CUDA_SAFE_CALL(hipMalloc((void **)&d_trace_buf, 
                TRACE_BUF_SIZE * sizeof(struct trace_content)));
    /*CUDA_SAFE_CALL(hipMalloc((void **)&d_pfilter, */
                /*sizeof(struct global_page_filter)));*/
    CUDA_SAFE_CALL(hipMalloc((void **)&d_result_queue,
                sizeof(struct global_race)));

    h_race_counter = 0;

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_race_counter), &h_race_counter, 
                sizeof(int)));
    printf("\nglobal timestamp queue size : %d\n", 
            MAX_PROCESS_NUM * sizeof(struct timestamp_queue));

}

extern "C" void module_cuda_update(
        /*struct global_history_queue *history, */
        /*struct timestamp_queue *h_gtq */
        /*struct global_page_filter *pfilter*/
        ); 

void module_cuda_update(
        /*struct global_history_queue *history, */
        /*struct timestamp_queue *h_gtq */
        /*struct global_page_filter *pfilter*/
        ) { 

    /*CUDA_SAFE_CALL(hipMemcpy(d_ghq, history,*/
                /*sizeof(struct global_history_queue),*/
                /*hipMemcpyHostToDevice));*/

    /*CUDA_SAFE_CALL(hipMemcpy(d_gtq, h_gtq,*/
                /*MAX_PROCESS_NUM * sizeof(struct timestamp_queue),*/
                /*hipMemcpyHostToDevice));*/

    /*CUDA_SAFE_CALL(hipMemcpy(d_pfilter, pfilter,*/
                /*sizeof(struct global_page_filter),*/
                /*hipMemcpyHostToDevice));*/

}

#ifdef COMPILE_TO_BINARY
int main(int argc, char** argv)  
{
    struct global_history_queue *ghq = NULL;
    struct global_page_filter *pfilter = NULL;

    /*struct timestamp_queue *h_gtq = NULL;*/
    struct trace_content *h_trace_buf = NULL;
    int *h_result_queue = NULL;

    module_cuda_init(ghq, pfilter);

    /*h_gtq = (struct timestamp_queue *)malloc(*/
            /*MAX_PROCESS_NUM * sizeof(struct timestamp_queue));*/
    /*memset(h_gtq, 0, MAX_PROCESS_NUM * sizeof(struct timestamp_queue));*/
    tool_global_timestamp_queue_init(h_gtq);

    /*history = (struct global_history_queue *)malloc(*/
            /*sizeof(struct global_history_queue));*/
    /*memset(history, 0, */
            /*sizeof(struct global_history_queue));*/
    tool_global_history_queue_init(ghq, MAX_LOAD_QUEUE_SIZE);

    /*pfilter = (struct global_page_filter *)malloc(*/
            /*sizeof(struct global_page_filter));*/
    /*memset(pfilter, 0, */
            /*sizeof(struct global_page_filter));*/
    tool_global_page_filter_init(pfilter, ghq, MAX_LOAD_QUEUE_SIZE);

    h_trace_buf = (struct trace_content *)malloc(
            TRACE_BUF_SIZE * sizeof(struct trace_content));
    memset(h_trace_buf, 0, TRACE_BUF_SIZE * sizeof(struct trace_content));
    tool_trace_buf_init(h_trace_buf, TRACE_BUF_SIZE);

    h_result_queue = (int *)malloc(sizeof(int) * TRACE_BUF_SIZE);
    memset(h_result_queue, 0, sizeof(int) * TRACE_BUF_SIZE);

    module_cuda_update(ghq, h_gtq, pfilter);
    module_cuda_stage_three(2, TRACE_BUF_SIZE, h_trace_buf);

    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(h_result_queue, d_result_queue,
                sizeof(struct global_race),
                hipMemcpyDeviceToHost));

    tool_result_queue_print(h_result_queue, TRACE_BUF_SIZE);

    module_cuda_free();

    free(h_gtq);
    /*free(history);*/
    /*free(pfilter);*/
    free(h_trace_buf);
    free(h_result_queue);

    hipDeviceReset();
    //cutilExit(argc, argv);
}
#endif

