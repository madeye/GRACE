#include <stdint.h>

#include <cutil_inline.h>

#include "../interface.h"
#include "data.h"
#include "match.h"
#include "match_kernel.cu"

int numThreads = 512;
int numBlocks = 512;

struct trace_content *d_trace_buf;

uint32_t old_index[MAX_PROCESS_NUM];

__host__ void module_cuda_tid_update_interface(int h_max_tid_num) {

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_max_tid_num), &h_max_tid_num, 
                sizeof(int)));
   
}

__host__ void module_cuda_config_register(int _numThreads)
{
    numThreads = _numThreads;
#if 1
    printf("\ncuda : thread number for each block : %d\n", numThreads);
#endif
}

__host__ void module_cuda_init_interface()
{
    hipSetDevice(cutGetMaxGflopsDeviceId());

    memset (old_index, 0, MAX_PROCESS_NUM * sizeof(uint32_t));

    cutilSafeCall(hipMalloc((void **)&d_trace_buf, 
                sizeof(struct trace_content) * TRACE_BUF_SIZE));
#if 1
    printf("\ncuda : global timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));
    printf("cuda : global history queue size : 0x%lx\n", 
            sizeof(struct global_history_queue));
    printf("cuda : global page filter size : 0x%lx\n", 
            sizeof(struct global_page_filter));
    printf("cuda : global race queue size : 0x%lx\n", 
            sizeof(struct global_race_queue));
#endif
}

__host__ void module_cuda_free_interface()
{
    cutilSafeCall(hipFree(d_trace_buf));
}

__host__ void module_cuda_global_timestamp_queue_update_interface(
        struct global_timestamp_queue *h_gts_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_gts_queue, 
                sizeof(struct global_timestamp_queue)));
}

__host__ void module_cuda_global_timestamp_queue_fetch_interface(
        struct global_timestamp_queue *h_gts_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_gts_queue, HIP_SYMBOL(gts), 
                sizeof(struct global_timestamp_queue)));
}

__host__ void module_cuda_timestamp_queue_update_interface(
        uint8_t tid, struct timestamp_queue *h_ts_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_ts_queue, 
                sizeof(struct timestamp_queue), 
                tid * sizeof(struct timestamp_queue), 
                hipMemcpyHostToDevice));
}

__host__ void module_cuda_timestamp_queue_fetch_interface(
        uint8_t tid, struct timestamp_queue *h_ts_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_ts_queue, HIP_SYMBOL(gts), 
                sizeof(struct timestamp_queue), 
                tid * sizeof(struct timestamp_queue), 
                hipMemcpyDeviceToHost));
}

/*__host__ void module_cuda_timestamp_entry_update_interface(*/
        /*uint8_t tid, uint32_t index, struct timestamp *h_ts_entry)*/
/*{*/
    /*cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_ts_entry,*/
                /*sizeof(struct timestamp),*/
                /*tid * sizeof(struct*/
                    /*timestamp_queue) +*/
                /*index * sizeof(struct timestamp),*/
                /*hipMemcpyHostToDevice));*/
/*}*/

#if 1
__host__ void module_cuda_timestamp_entry_update_interface(
        uint8_t max_tid_num, uint32_t *ctx, struct timestamp_queue *h_ts_queue)
{
    int i;
    for (i = 1; i < max_tid_num; i++) {
    /*for (i = 0; i < MAX_PROCESS_NUM; i++) {*/

        /*cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry, */
                    /*(MAX_TIMESTAMP_NUM) * sizeof(struct timestamp), */
                    /*i * sizeof(struct timestamp_queue), */
                    /*hipMemcpyHostToDevice));*/

        if (ctx[i] > old_index[i]) {
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[old_index[i]], 
                        (ctx[i] - old_index[i] + 1) * sizeof(struct timestamp), 
                        i * sizeof(struct timestamp_queue) + 
                        old_index[i] * sizeof(struct timestamp), 
                        hipMemcpyHostToDevice));
        } else if (ctx[i] < old_index[i]) {
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[old_index[i]], 
                        (MAX_TIMESTAMP_NUM - old_index[i] - 1) * sizeof(struct timestamp), 
                        i * sizeof(struct timestamp_queue) + 
                        old_index[i] * sizeof(struct timestamp), 
                        hipMemcpyHostToDevice));
            cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), &h_ts_queue[i].entry[0], 
                        (ctx[i] - 0 + 1) * sizeof(struct timestamp), 
                        i * sizeof(struct timestamp_queue) + 
                        0,
                        hipMemcpyHostToDevice));
        }
        old_index[i] = ctx[i];
    }

}

#endif

__host__ void module_cuda_timestamp_entry_fetch_interface(
        uint8_t tid, uint32_t index, struct timestamp *h_ts_entry)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_ts_entry, HIP_SYMBOL(gts), 
                sizeof(struct timestamp), 
                tid * sizeof(struct timestamp_queue) + 
                index * sizeof(struct timestamp), 
                hipMemcpyDeviceToHost));
}

__host__ void module_cuda_global_history_queue_update_interface(
        struct global_history_queue *h_gh_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gh), h_gh_queue, 
                sizeof(struct global_history_queue)));
}

__host__ void module_cuda_global_history_queue_fetch_interface(
        struct global_history_queue *h_gh_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_gh_queue, HIP_SYMBOL(gh), 
                sizeof(struct global_history_queue)));
}

__host__ void module_cuda_history_hash_queue_update_interface(
        uint8_t tid, struct history_hash_queue *h_hh_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gh), h_hh_queue, 
                sizeof(struct history_hash_queue), 
                tid * sizeof(struct history_hash_queue), 
                hipMemcpyHostToDevice));
}

__host__ void module_cuda_history_hash_queue_fetch_interface(
        uint8_t tid, struct history_hash_queue *h_hh_queue)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_hh_queue, HIP_SYMBOL(gh), 
                sizeof(struct history_hash_queue), 
                tid * sizeof(struct history_hash_queue), 
                hipMemcpyDeviceToHost));
}

__host__ void module_cuda_global_page_filter_update_interface(
        struct global_page_filter *h_gpf)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gpf), h_gpf, 
                sizeof(struct global_page_filter)));
}

__host__ void module_cuda_global_page_filter_fetch_interface(
        struct global_page_filter *h_gpf)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_gpf, HIP_SYMBOL(gpf), 
                sizeof(struct global_page_filter)));
}

__host__ void module_cuda_page_filter_update_interface(
        uint8_t tid, struct page_filter *h_pf)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gpf), h_pf, 
                sizeof(struct page_filter), 
                tid * sizeof(struct page_filter), 
                hipMemcpyHostToDevice));
}

__host__ void module_cuda_page_filter_fetch_interface(
        uint8_t tid, struct page_filter *h_pf)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_pf, HIP_SYMBOL(gpf), 
                sizeof(struct page_filter), 
                tid * sizeof(struct page_filter), 
                hipMemcpyDeviceToHost));
}

__host__ void module_cuda_global_race_queue_fetch_interface(
        struct global_race_queue *h_gr)
{
    cutilSafeCall(hipMemcpyFromSymbol(h_gr, HIP_SYMBOL(gr), 
                sizeof(struct global_race_queue)));
#if 1
    printf("cuda : race : %d\n", h_gr->sum);
#endif
}

__host__ void module_cuda_match_with_trace_buf_interface(
        uint8_t tid, uint32_t size, struct trace_content *h_trace_buf)
{
    cutilSafeCall(hipMemcpy(d_trace_buf, h_trace_buf, 
                sizeof(struct trace_content) * size, 
                hipMemcpyHostToDevice));

    numBlocks = (size + numThreads - 1) / numThreads;

    module_match_with_trace_buf_on_cuda<<<numBlocks, numThreads>>>(size,
            d_trace_buf);
}

