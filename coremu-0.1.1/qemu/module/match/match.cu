#include <stdint.h>

#include <cutil_inline.h>

#include "data.h"
#include "match.h"
#include "match_kernel.cu"

#define TRACE_BUF_SIZE 1
#define HISTORY_QUEUE_SIZE 128

#define NUM_BLOCKS  1
#define NUM_THREADS HISTORY_QUEUE_SIZE

//#define COMPILE_TO_BINARY
#ifdef COMPILE_TO_BINARY
__host__ static void tool_global_timestamp_queue_init(
        struct global_timestamp_queue *gts_queue)
{
    gts_queue->thread[0].entry[0].scalar[0] = 0;
    gts_queue->thread[0].entry[0].scalar[1] = 0;
    gts_queue->thread[0].entry[1].scalar[0] = 1;
    gts_queue->thread[0].entry[1].scalar[1] = 1;
    gts_queue->thread[0].entry[2].scalar[0] = 2;
    gts_queue->thread[0].entry[2].scalar[1] = 1;
    gts_queue->thread[0].entry[3].scalar[0] = 3;
    gts_queue->thread[0].entry[3].scalar[1] = 3;
    gts_queue->thread[0].entry[4].scalar[0] = 4;
    gts_queue->thread[0].entry[4].scalar[1] = 4;

    gts_queue->thread[1].entry[0].scalar[0] = 0;
    gts_queue->thread[1].entry[0].scalar[1] = 0;
    gts_queue->thread[1].entry[1].scalar[0] = 0;
    gts_queue->thread[1].entry[1].scalar[1] = 1;
    gts_queue->thread[1].entry[2].scalar[0] = 2;
    gts_queue->thread[1].entry[2].scalar[1] = 2;
    gts_queue->thread[1].entry[3].scalar[0] = 3;
    gts_queue->thread[1].entry[3].scalar[1] = 3;
    gts_queue->thread[1].entry[4].scalar[0] = 4;
    gts_queue->thread[1].entry[4].scalar[1] = 4;
}

#define STRIDE 4

__host__ static void tool_history_queue_init(
        struct history_entry *history_queue, int size)
{
    int i;

    printf("\nHistory Queue Entry :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        history_queue[i].content.address = (i % STRIDE) << 2;
        history_queue[i].content.tid = 1;
        history_queue[i].content.type = 2;
        history_queue[i].content.size = 4;
        history_queue[i].content.index = (i / STRIDE);
        history_queue[i].content.pc = 0xf;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                history_queue[i].content.address, history_queue[i].content.tid,
                history_queue[i].content.type, history_queue[i].content.size,
                history_queue[i].content.index, history_queue[i].content.pc);
    }
}

__host__ static void tool_trace_buf_init(
        struct trace_content *trace_buf, int size)
{
    int i;

    printf("\nTrace Queue Content :\n");
    printf("No\tAddress\tTid\tType\tSize\tIndex\tPC\n");

    for (i = 0; i < size; i++)
    {
        trace_buf[i].address = (STRIDE >> 1) << 2;
        trace_buf[i].tid = 0;
        trace_buf[i].type = 1;
        trace_buf[i].size = 4;
        trace_buf[i].index = 1;
        trace_buf[i].pc = 0x4;

        printf("%d\t0x%llx\t%d\t%d\t%d\t%d\t0x%x\n", i, 
                trace_buf[i].address, trace_buf[i].tid, 
                trace_buf[i].type, trace_buf[i].size, 
                trace_buf[i].index, trace_buf[i].pc);
    }
}

__host__ static void tool_result_queue_print(
        int *result_queue, int size)
{
    int i;

    printf("\nResult Queue Entry :\n");
    printf("No\tResult\n");

    for (i = 0; i < size; i++)
    {
        printf("%d\t%d\n", i, result_queue[i]);
    }
}
#endif

struct history_entry *d_history_queue;
struct trace_content *d_trace_buf;
int *d_result_queue;

__host__ void module_cuda_init_interface()
{
    cutilSafeCall(hipMalloc((void **)&d_history_queue, 
                sizeof(struct history_entry) * HISTORY_QUEUE_SIZE));
    cutilSafeCall(hipMalloc((void **)&d_trace_buf, 
                sizeof(struct trace_content)));
    cutilSafeCall(hipMalloc((void **)&d_result_queue,
                sizeof(int) * HISTORY_QUEUE_SIZE));

    hipSetDevice(cutGetMaxGflopsDeviceId());

    printf("\nglobal timestamp queue size : 0x%lx\n", 
            sizeof(struct global_timestamp_queue));
}

__host__ void module_cuda_free_interface()
{
    cutilSafeCall(hipFree(d_history_queue));
    cutilSafeCall(hipFree(d_trace_buf));
    cutilSafeCall(hipFree(d_result_queue));
}

__host__ void module_cuda_timestamp_update_interface(
        struct global_timestamp_queue *h_gts_queue)
{
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gts), h_gts_queue, 
                sizeof(struct global_timestamp_queue)));
}

__host__ void module_cuda_match_interface(
        struct history_entry *h_history_queue,
        struct trace_content *h_trace_buf,
        int *h_result_queue)
{
    cutilSafeCall(hipMemcpy(d_history_queue, h_history_queue, 
                sizeof(struct history_entry) * HISTORY_QUEUE_SIZE, 
                hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_trace_buf, h_trace_buf, 
                sizeof(struct trace_content), 
                hipMemcpyHostToDevice));

    module_match_on_cuda<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_history_queue, d_trace_buf, d_result_queue);

    cutilSafeCall(hipMemcpy(h_result_queue, d_result_queue, 
                sizeof(int) * HISTORY_QUEUE_SIZE, 
                hipMemcpyDeviceToHost));
}

#ifdef COMPILE_TO_BINARY
int main(int argc, char** argv)  
{
    struct global_timestamp_queue *h_gts_queue;
    struct history_entry *h_history_queue;
    struct trace_content *h_trace_buf;
    int *h_result_queue;

    h_gts_queue = (struct global_timestamp_queue *)malloc(
            sizeof(struct global_timestamp_queue));
    memset(h_gts_queue, 0, sizeof(struct global_timestamp_queue));
    tool_global_timestamp_queue_init(h_gts_queue);

    h_history_queue = (struct history_entry *)malloc(
            sizeof(struct history_entry) * HISTORY_QUEUE_SIZE);
    memset(h_history_queue, 0, 
            sizeof(struct history_entry) * HISTORY_QUEUE_SIZE);
    tool_history_queue_init(h_history_queue, HISTORY_QUEUE_SIZE);

    h_trace_buf = (struct trace_content *)malloc(
            sizeof(struct trace_content));
    memset(h_trace_buf, 0, sizeof(struct trace_content));
    tool_trace_buf_init(h_trace_buf, TRACE_BUF_SIZE);

    h_result_queue = (int *)malloc(sizeof(int) * HISTORY_QUEUE_SIZE);
    memset(h_result_queue, 0, sizeof(int) * HISTORY_QUEUE_SIZE);

    module_cuda_init_interface();
    module_cuda_timestamp_update_interface(h_gts_queue);
    module_cuda_match_interface(h_history_queue, h_trace_buf, h_result_queue);
    module_cuda_free_interface();

    tool_result_queue_print(h_result_queue, HISTORY_QUEUE_SIZE);

    free(h_history_queue);
    free(h_trace_buf);
    free(h_result_queue);

    hipDeviceReset();
    //cutilExit(argc, argv);
}
#endif

