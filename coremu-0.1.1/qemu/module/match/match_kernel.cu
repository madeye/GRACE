#include "hip/hip_runtime.h"
#ifndef _HISTORY_KERNEL_H_
#define _HISTORY_KERNEL_H_

__constant__ struct global_timestamp_queue gts;

__device__ static inline int module_timestamp_order_on_cuda(
        uint8_t tid1, uint32_t index1, uint8_t tid2, uint32_t index2) 
{
    const struct timestamp *ts1 = &gts.thread[tid1].entry[index1];
    const struct timestamp *ts2 = &gts.thread[tid2].entry[index2];

    return (((ts1->scalar[tid1] < ts2->scalar[tid1]) && 
                (ts1->scalar[tid2] < ts2->scalar[tid2])) || 
            ((ts1->scalar[tid1] > ts2->scalar[tid1]) && 
             (ts1->scalar[tid2] > ts2->scalar[tid2])));
}

__global__ static void module_match_on_cuda(
        struct history_entry *history_queue, 
        struct trace_content *trace_buf, 
        int *result_queue)
{
    const int x = threadIdx.x;

    struct trace_content *temp_entry = &history_queue[x].content;
    const int other_tid = temp_entry->tid;
    const int other_index = temp_entry->index;
    const int other_address = temp_entry->address;

    struct trace_content *temp_content = &trace_buf[0];
    const int self_tid = temp_content->tid;
    const int self_index = temp_content->index;
    const int self_address = temp_content->address;

#if 0
    result_queue[x] = (self_address == other_address) &&
        !module_timestamp_order_on_cuda(other_tid, other_index, self_tid, self_index);
#else
    result_queue[x] = 1;
#endif
}

#endif /* _HISTORY_KERNEL_H_ */
